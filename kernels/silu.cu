#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <stdint.h>

__device__ __forceinline__ float expg(float a) { return expf(a); }
__device__ __forceinline__ __half expg(__half a) { return hexp(a); }
__device__ __forceinline__ __hip_bfloat16 expg(__hip_bfloat16 a) { return hexp(a); }

template<typename scalar_t>
inline __device__ scalar_t silu(
  scalar_t __restrict__ x)
{
  return x / (static_cast<scalar_t>(1) + expg(-x));
}

template<typename scalar_t>
__global__ void silu_kernel(
  scalar_t* __restrict__ x_ptr,
  scalar_t* __restrict__ out_ptr,
  const int numel) {

  for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
    out_ptr[i] = silu<scalar_t>(x_ptr[i]);
  }
}

#define CALL_SILU(T)                                                          \
  silu_kernel<T><<<grid, block, 0, stream>>>(                                 \
  reinterpret_cast<T*>(x),                                                    \
  reinterpret_cast<T*>(out),                                                  \
  numel);

extern "C" void silu(
  void *x,
  void *out,

  int32_t num_blocks,
  int32_t num_threads,
  int32_t numel,

  uint32_t dtype // 0 => f16; 1 => bf16; 2 => f32
  ) {
  dim3 grid(num_blocks);
  dim3 block(num_threads);
  const hipStream_t stream = 0;

  if (dtype == 0){
    CALL_SILU(half);
  } else if (dtype == 1) {
    CALL_SILU(__hip_bfloat16);
  } else if (dtype == 2) {
    CALL_SILU(float);
  }
}
